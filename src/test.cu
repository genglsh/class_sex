#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "hdf5.h"
// #include <opencv2/opencv.hpp>
// using namespace cv;
using namespace std;
#include <TooN/TooN.h>
#include <TooN/se3.h>
#include <TooN/GR_SVD.h>

__device__ int get_pos(){
    int 
}

struct Matrix4 {
    float4 data[4];

    inline __host__ __device__ float3 get_translation() const {
        return make_float3(data[0].w, data[1].w, data[2].w);
    }
};
inline Matrix4 getCameraMatrix( const float4 & k ){
    Matrix4 K;
    K.data[0] = make_float4(k.x, 0, k.z, 0);
    K.data[1] = make_float4(0, k.y, k.w, 0);
    K.data[2] = make_float4(0, 0, 1, 0);
    K.data[3] = make_float4(0, 0, 0, 1);
    return K;
}

inline Matrix4 getInverseCameraMatrix( const float4 & k ){
    Matrix4 invK;
    invK.data[0] = make_float4(1.0f/k.x, 0, -k.z/k.x, 0);
    invK.data[1] = make_float4(0, 1.0f/k.y, -k.w/k.y, 0);
    invK.data[2] = make_float4(0, 0, 1, 0);
    invK.data[3] = make_float4(0, 0, 0, 1);
    return invK;
}


std::ostream & operator<<( std::ostream & out, const Matrix4 & m ){
    for(unsigned i = 0; i < 4; ++i)
        out << m.data[i].x << "  " << m.data[i].y << "  " << m.data[i].z << "  " << m.data[i].w << "\n";
    return out;
}
 
__global__ void point2depthimage(Matrix4 r_inv, float* depth_image, float* point_sequence) {
    
	printf("aaa\n");
}
 
int main(void) {
	mykernel<<<2,2>>>();
    printf("Hello World!\n");
    hipDeviceSynchronize();
    hipDeviceReset();
}
 