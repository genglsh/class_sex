#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
// #include <opencv2/opencv.hpp>
#include<fstream>
// #include <TooN/TooN.h>
// #include <TooN/se3.h>
// #include <TooN/GR_SVD.h>

// using namespace cv;
using namespace std;

__device__ int get_pos(){
    return threadIdx.x + blockIdx.x * blockDim.x;
    
}
struct Img{
    int* data;
    int image_width;
    int image_height;
    int img_arr_size;
    Img(){

    }
    Img(int w, int h){
        image_height = h;
        image_width = w;
        img_arr_size = sizeof(int) * image_width*image_height;
        hipMalloc((void**)&data, img_arr_size);
    }
};
struct Matrix4 {
    float4 data[4];

    inline __host__ __device__ float3 get_translation() const {
        return make_float3(data[0].w, data[1].w, data[2].w);
    }
};
inline Matrix4 getCameraMatrix( const float4 & k ){
    Matrix4 K;
    K.data[0] = make_float4(k.x, 0, k.z, 0);
    K.data[1] = make_float4(0, k.y, k.w, 0);
    K.data[2] = make_float4(0, 0, 1, 0);
    K.data[3] = make_float4(0, 0, 0, 1);
    return K;
}

inline Matrix4 getInverseCameraMatrix( const float4 & k ){
    Matrix4 invK;
    invK.data[0] = make_float4(1.0f/k.x, 0, -k.z/k.x, 0);
    invK.data[1] = make_float4(0, 1.0f/k.y, -k.w/k.y, 0);
    invK.data[2] = make_float4(0, 0, 1, 0);
    invK.data[3] = make_float4(0, 0, 0, 1);
    return invK;
}

inline __host__ __device__ float3 make_float3(float4 a)
{
    // printf("make_float3_start\n");
    return make_float3(a.x, a.y, a.z);
}

inline __host__ __device__ float dot(float3 a, float3 b)
{
    // printf("dot_start\n");
    return a.x * b.x + a.y * b.y + a.z * b.z;
}


inline __host__ __device__ float3 rotate( const Matrix4 & M, const float3 & v){
    // printf("start\n");
    float3 ans = make_float3(dot(make_float3(M.data[0]), v),dot(make_float3(M.data[1]), v),dot(make_float3(M.data[2]), v));
        // printf("end\n");
    return ans;
}



std::ostream & operator<<( std::ostream & out, const Matrix4 & m ){
    for(unsigned i = 0; i < 4; ++i)
        out << m.data[i].x << "  " << m.data[i].y << "  " << m.data[i].z << "  " << m.data[i].w << "\n";
    return out;
}
 
// __global__ void point2depthimage(Matrix4 r_inv, float* depth_image, float* point_sequence) {
    
// 	printf("aaa\n");
// }
__global__ void mykernel(float3* a, const Matrix4 r_inv, int max_num,Img* img){
    // printf("%d %d %d %d \n", r_inv.data[0].x, r_inv.data[0].y, r_inv.data[0].z, r_inv.data[0].w);
    
    int index = get_pos();
    if(index >= max_num)
        return ;
    // printf("index is %d x is %f, y is %f, z is %f\n", index,arr[index].x, arr[index].y, arr[index].z);
    float3 pixel_  = rotate(r_inv, a[index]);
    int depth = int(pixel_.z);
    int2 image_pos =make_int2(pixel_.x / pixel_.z, pixel_.y / pixel_.z);
    if(image_pos.x >= img->image_width || image_pos.y >= img->image_height )
        return;
    else{
        img->data[img->image_width * image_pos.y + image_pos.x] = depth;
    }
    printf("index is %d,x is %f, y is %f, z is %f, image_pos.x is %d, image_pos.z is %d\n",index,pixel_.x, pixel_.y, pixel_.z, image_pos.x, image_pos.y);
}

int num = 1000;
float3* simulation_array;
float3* test_gpu_array;
int* img;
int array_size = sizeof(float3)*num;
float4 four_element = make_float4(1,1,1,1);
int img_width = 100, img_height = 200;
int* image_show;
Img* img_show = new Img(img_width, img_height);

// Matrix4 camera_M = getCameraMatrix(rotate_matrix);
Matrix4 inv_camera_M = getInverseCameraMatrix(four_element);

int main() {
    simulation_array = (float3*)malloc(array_size);
    image_show = (int*)malloc(img_show->img_arr_size);

    // mykernel<<<100,1000>>>();
    for(int x = 0; x<num;x++){
        simulation_array[x] = make_float3(rand()%10,rand()%10,rand()%10);
        // printf("%f, %f, %f\n", simulation_array[x].x, simulation_arr/ay[x].y,simulation_array[x].z);
    }
    hipMalloc((void**)&test_gpu_array, array_size);
    hipMemcpy(test_gpu_array,simulation_array,array_size, hipMemcpyHostToDevice); 
    // cout<<inv_camera_M;
    mykernel<<<100,10>>>(test_gpu_array, inv_camera_M, num, img_show);
    hipMemcpy(image_show,img_show->data, img_show->img_arr_size, hipMemcpyDeviceToHost); 
    
    // printf("Hello World!\n");
    hipDeviceSynchronize();

    ofstream ofile;               
    ofile.open("image_show.txt");     
    for(int i=0; i<img_height;i++)
        for(int j=0;j<img_width;j++){
            if(j == img_width-1)
                ofile<<image_show[i*img_width + j]<<'\n';
            else
                ofile<<image_show[i*img_width + j]<<' ';   
        }
    ofile.close();               

    hipDeviceReset();
    // uchar *ptmp = NULL;
    // Mat depth_image = Mat(img_height, img_width, CV_8UC1);
    // for(int i = 0 ;i<img_height;i++){
    //     // ptmp = depth_image.ptr<uchar>(i);
    //     for(int ii = 0; ii < img_width; ii++){
    //         depth_image.at<uchar>(i,ii) = image_show[ii+i*img_show->image_width];
    //     }
    // }
    // // namedWindow("depth_image");
    // imshow("depth_image", depth_image);
    // waitKey(0);
}
 